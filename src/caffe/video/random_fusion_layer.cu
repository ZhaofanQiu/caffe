#include "hip/hip_runtime.h"

/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth�� Zhaofan Qiu
** mail�� zhaofanqiu@gmail.com
** date�� 2015/12/13
** desc�� RandomFusion layer
*********************************************************************************/

#include <string>
#include <utility>
#include <vector>

#include "caffe/video/random_fusion_layer.hpp"

namespace caffe {

template <typename Dtype>
void RandomFusionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
	Dtype* top_data = top[0]->mutable_gpu_data();
	const int count = top[0]->count();

	caffe_gpu_set(count, Dtype(0.), top_data);
	if (this->phase_ == TRAIN)
	{
		switch (this->layer_param().random_fusion_param().random())
		{
		case caffe::RandomFusionParameter_RandomMethod_Gaussion:
		  caffe::caffe_rng_gaussian(bottom.size(), (Dtype)mean_, (Dtype)std_, &random_vec_[0]);
		  for (int i = 0; i < bottom.size(); i++)
		  {
			  random_vec_[i] = std::max(random_vec_[i], (Dtype)0.);
		  }
		  break;
		case caffe::RandomFusionParameter_RandomMethod_Bernoulli:
			caffe::caffe_rng_bernoulli(bottom.size(), (Dtype)prob_, &random_idx_[0]);
			for (int i = 0; i < bottom.size(); i++)
			{
				random_vec_[i] = static_cast<Dtype>(random_idx_[i]);
			}
			break;
		default:
			LOG(FATAL) << "Unknown random operation.";
		}
		Dtype sum = static_cast<Dtype>(0.);
		for (int i = 0; i < bottom.size(); i++)
		{
			sum += random_vec_[i];
		}
		if (sum < 1e-6)
		{
			for (int i = 0; i < bottom.size(); i++)
			{
				random_vec_[i] = static_cast<Dtype>(1.0 / bottom.size());
			}
		}
		else
		{
			for (int i = 0; i < bottom.size(); i++)
			{
				random_vec_[i] /= sum;
			}
		}
	}
	else
	{
		for (int i = 0; i < bottom.size(); i++)
		{
			random_vec_[i] = static_cast<Dtype>(1.0 / bottom.size());
		}
	}
	for (int i = 0; i < bottom.size(); ++i) {
		caffe_gpu_axpy(count, random_vec_[i], bottom[i]->gpu_data(), top_data);
	}
}

template <typename Dtype>
void RandomFusionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
	const int count = top[0]->count();
	const Dtype* top_data = top[0]->gpu_data();
	const Dtype* top_diff = top[0]->gpu_diff();
	for (int i = 0; i < bottom.size(); ++i) {
		Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
		if (propagate_down[i])
		{
			caffe_gpu_scale(count, random_vec_[i], top_diff, bottom_diff);
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(RandomFusionLayer);


}  // namespace caffe
