#include "hip/hip_runtime.h"
﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/12/13
** desc： L1Loss layer
*********************************************************************************/

#include <string>
#include <utility>
#include <vector>

#include "caffe/video/l1_loss_layer.hpp"

namespace caffe {
	template <typename Dtype>
	void L1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int count = bottom[0]->count();
		const Dtype* bottom_data = bottom[0]->gpu_data();

		Dtype loss;
		caffe_gpu_asum(count, bottom_data, &loss);
		loss /= bottom[0]->shape(0);
		top[0]->mutable_cpu_data()[0] = loss;
	}

	template <typename Dtype>
	__global__ void L1BackKernel(const int nthreads,
		const Dtype* X, const Dtype eps, Dtype* Y) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			Y[index] = (X[index] > eps) - (X[index] < -eps);
		}
	}

	template <typename Dtype>
	void L1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const int count = bottom[0]->count();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

		L1BackKernel<Dtype> << <CAFFE_GET_BLOCKS(count),
			CAFFE_CUDA_NUM_THREADS >> >(count, bottom_data, this->eps_, bottom_diff);
		caffe_gpu_scal(count, top[0]->cpu_diff()[0], bottom_diff);
	}

	INSTANTIATE_LAYER_GPU_FUNCS(L1LossLayer);

}  // namespace caffe
