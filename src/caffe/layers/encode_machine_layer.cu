#include "hip/hip_runtime.h"
﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/12/2
** desc： EncodeMachineLayer layer
*********************************************************************************/

#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/vol2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/video_layers.hpp"

namespace caffe {
	template <typename Dtype>
	__global__ void SampleAct(const int count, const Dtype* input, const Dtype* noise,
		Dtype* output) {
		CUDA_KERNEL_LOOP(index, count) {
			const Dtype t = input[index] + noise[index] * sqrt(1. / (1 + exp(-input[index])));
			output[index] = t * (t > 0);
		}
	}

template <typename Dtype>
void EncodeMachineLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	// share parameters
	vector<shared_ptr<Blob<Dtype> > > net_params = net_->params();
	for (int i = 0; i < net_params.size(); ++i)
	{
		net_params[i]->ShareData(*blobs_[i]);
		net_params[i]->ShareDiff(*blobs_[i]);
	}

	// copy bottom[0] -> v0_, bottom[0] -> vis_blob_
	caffe_copy(count_v_, bottom[0]->gpu_data(), v0_->mutable_gpu_data());
	caffe_copy(count_v_, bottom[0]->gpu_data(), vis_blob_->mutable_gpu_data());

	// init encode, h0_ -> top[0]
	net_->ForwardFromTo(encode_begin_, encode_end_);
	caffe_copy(count_h_, hid_blob_->gpu_data(), top[0]->mutable_gpu_data());
	caffe_copy(count_h_, hid_blob_->gpu_data(), mean_h0_->mutable_gpu_data());

	// sample hid_blob_
	caffe_gpu_rng_gaussian<Dtype>(count_h_, 0, 1, sample_h_->mutable_gpu_data());
	SampleAct<Dtype> << <CAFFE_GET_BLOCKS(count_h_), CAFFE_CUDA_NUM_THREADS >> >(
		count_h_, hid_blob_->gpu_data(), sample_h_->gpu_data(), hid_blob_->mutable_gpu_data());

	// 1~cd_k decode/encode
	for (int k = 0; k < cd_k_; k++)
	{
		// decode hk-1 -> vk
		net_->ForwardFromTo(decode_begin_, decode_end_);
		// sample vk -> vk
		caffe_gpu_rng_gaussian<Dtype>(count_v_, 0, 1, sample_v_->mutable_gpu_data());
		SampleAct<Dtype> << <CAFFE_GET_BLOCKS(count_v_), CAFFE_CUDA_NUM_THREADS >> >(
			count_v_, re_vis_blob_->gpu_data(), sample_v_->gpu_data(), re_vis_blob_->mutable_gpu_data());

		if (k == 0)
		{
			// calculate loss
			caffe_gpu_sub(count_v_, v0_->gpu_data(), vis_blob_->gpu_data(), diff_v_->mutable_gpu_data());
			Dtype dot;
			caffe_gpu_dot(count_v_, diff_v_->gpu_data(), diff_v_->gpu_data(), &dot);
			Dtype loss = loss_weight_ * dot / bottom[0]->shape(0) / Dtype(2);
			top[1]->mutable_cpu_data()[0] = loss;
		}

		// encode vk -> hk
		caffe_copy(count_v_, re_vis_blob_->gpu_data(), vis_blob_->mutable_gpu_data());
		net_->ForwardFromTo(encode_begin_, encode_end_);
		// sample hk -> hk
		caffe_gpu_rng_gaussian<Dtype>(count_h_, 0, 1, sample_h_->mutable_gpu_data());
		SampleAct<Dtype> << <CAFFE_GET_BLOCKS(count_h_), CAFFE_CUDA_NUM_THREADS >> >(
			count_h_, hid_blob_->gpu_data(), sample_h_->gpu_data(), hid_blob_->mutable_gpu_data());
	}
	caffe_copy(count_h_, hid_blob_->gpu_data(), hk_->mutable_gpu_data());
	caffe_copy(count_v_, vis_blob_->gpu_data(), vk_->mutable_gpu_data());

}

template <typename Dtype>
void EncodeMachineLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	// share parameters
	vector<shared_ptr<Blob<Dtype> > > net_params = net_->params();
	for (int i = 0; i < net_params.size(); ++i)
	{
		net_params[i]->ShareData(*blobs_[i]);
		net_params[i]->ShareDiff(*blobs_[i]);
	}

	if (this->phase_ == TRAIN)
	{
		// Grad from top
		caffe_copy(count_v_, bottom[0]->gpu_data(), vis_blob_->mutable_gpu_data());
		net_->ForwardFromTo(encode_begin_, encode_end_);

		caffe_copy(count_h_, top[0]->gpu_diff(), hid_blob_->mutable_gpu_diff());
		net_->BackwardFromTo(encode_end_, encode_begin_);

		// E<p(h, v)> 1
		caffe_copy(count_h_, hk_->gpu_data(), hid_blob_->mutable_gpu_data());
		net_->ForwardFromTo(decode_begin_, decode_end_);

		caffe_gpu_set(count_v_, Dtype(0.), re_vis_blob_->mutable_gpu_diff());
		caffe_gpu_axpy(count_v_, loss_weight_, vk_->gpu_data(), re_vis_blob_->mutable_gpu_diff());
		net_->BackwardFromTo(decode_end_, decode_begin_);

		// E<p(h, v)> 2
		caffe_copy(count_v_, vk_->gpu_data(), vis_blob_->mutable_gpu_data());
		net_->ForwardFromTo(encode_begin_, encode_end_);

		caffe_gpu_set(count_h_, Dtype(0.), hid_blob_->mutable_gpu_diff());
		caffe_gpu_axpy(count_h_, loss_weight_, hk_->gpu_data(), hid_blob_->mutable_gpu_diff());
		net_->BackwardFromTo(encode_end_, encode_begin_);
	}

	if (s_k_ == 0)
	{
		// sample h0
		caffe_gpu_set(count_h_, Dtype(0.), sample_h_->mutable_gpu_data());
		SampleAct<Dtype> << <CAFFE_GET_BLOCKS(count_h_), CAFFE_CUDA_NUM_THREADS >> >(
			count_h_, mean_h0_->gpu_data(), sample_h_->gpu_data(), h0_->mutable_gpu_data());

		// E<p(h|v)> 1
		caffe_copy(count_h_, h0_->gpu_data(), hid_blob_->mutable_gpu_data());
		net_->ForwardFromTo(decode_begin_, decode_end_);

		caffe_gpu_set(count_v_, Dtype(0.), re_vis_blob_->mutable_gpu_diff());
		caffe_gpu_axpy(count_v_, -loss_weight_, v0_->gpu_data(), re_vis_blob_->mutable_gpu_diff());
		net_->BackwardFromTo(decode_end_, decode_begin_);
		// E<p(h|v)> 2
		caffe_copy(count_v_, v0_->gpu_data(), vis_blob_->mutable_gpu_data());
		net_->ForwardFromTo(encode_begin_, encode_end_);

		caffe_gpu_set(count_h_, Dtype(0.), hid_blob_->mutable_gpu_diff());
		caffe_gpu_axpy(count_h_, -loss_weight_, h0_->gpu_data(), hid_blob_->mutable_gpu_diff());
		net_->BackwardFromTo(encode_end_, encode_begin_);
	}
	else
	{
		for (int k = 0; k < s_k_; ++k)
		{
			// sample h0
			caffe_gpu_rng_gaussian<Dtype>(count_h_, 0, 1, sample_h_->mutable_gpu_data());
			SampleAct<Dtype> << <CAFFE_GET_BLOCKS(count_h_), CAFFE_CUDA_NUM_THREADS >> >(
				count_h_, mean_h0_->gpu_data(), sample_h_->gpu_data(), h0_->mutable_gpu_data());

			// E<p(h|v)> 1
			caffe_copy(count_h_, h0_->gpu_data(), hid_blob_->mutable_gpu_data());
			net_->ForwardFromTo(decode_begin_, decode_end_);

			caffe_gpu_set(count_v_, Dtype(0.), re_vis_blob_->mutable_gpu_diff());
			caffe_gpu_axpy(count_v_, -loss_weight_ / s_k_, v0_->gpu_data(), re_vis_blob_->mutable_gpu_diff());
			net_->BackwardFromTo(decode_end_, decode_begin_);
			// E<p(h|v)> 2
			caffe_copy(count_v_, v0_->gpu_data(), vis_blob_->mutable_gpu_data());
			net_->ForwardFromTo(encode_begin_, encode_end_);

			caffe_gpu_set(count_h_, Dtype(0.), hid_blob_->mutable_gpu_diff());
			caffe_gpu_axpy(count_h_, -loss_weight_ / s_k_, h0_->gpu_data(), hid_blob_->mutable_gpu_diff());
			net_->BackwardFromTo(encode_end_, encode_begin_);
		}
	}

	CUDA_POST_KERNEL_CHECK;
}
INSTANTIATE_LAYER_GPU_FUNCS(EncodeMachineLayer);

}  // namespace caffe
