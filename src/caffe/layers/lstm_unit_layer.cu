#include "hip/hip_runtime.h"
﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/9/11
** desc： LSTMUnit layer
*********************************************************************************/

#include <algorithm>
#include <cmath>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/video_layers.hpp"

namespace caffe {

	template <typename Dtype>
	__device__ Dtype sigmoid(const Dtype x) {
		return Dtype(1) / (Dtype(1) + exp(-x));
	}

	/*
	template <typename Dtype>
	__device__ Dtype tanh(const Dtype x) {
		return Dtype(2) * sigmoid(Dtype(2) * x) - Dtype(1);
	}
	*/

	template <typename Dtype>
	__global__ void LSTMActsForward(const int nthreads, const int dim,
		const Dtype* X, Dtype* X_acts) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int x_dim = 4 * dim;
			const int d = index % x_dim;
			if (d < 3 * dim) {
				X_acts[index] = sigmoid(X[index]);
			}
			else {
				X_acts[index] = tanh(X[index]);
			}
		}
	}

	template <typename Dtype>
	__global__ void LSTMUnitForward(const int nthreads, const int dim,
		const Dtype* C_prev, const Dtype* X, 
		Dtype* C, Dtype* H) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int n = index / dim;
			const int d = index % dim;
			const Dtype* X_offset = X + 4 * dim * n;
			const Dtype i = X_offset[d];
			const Dtype f = X_offset[1 * dim + d];
			const Dtype o = X_offset[2 * dim + d];
			const Dtype g = X_offset[3 * dim + d];
			const Dtype c_prev = C_prev[index];
			const Dtype c = f * c_prev + i * g;
			C[index] = c;
			const Dtype tanh_c = tanh(c);
			H[index] = o * tanh_c;
		}
	}

	template <typename Dtype>
	void LSTMUnitLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int count = top[1]->count();
		const int X_count = bottom[1]->count();
		for (int i = 0; i < this->input_num_; i++)
		{
			const Dtype* C_prev = bottom[i * 2]->gpu_data();
			const Dtype* X = bottom[i * 2 + 1]->gpu_data();
			Dtype* C = top[i * 2]->mutable_gpu_data();
			Dtype* H = top[i * 2 + 1]->mutable_gpu_data();
			Dtype* X_acts = X_acts_[i]->mutable_gpu_data();
			// NOLINT_NEXT_LINE(whitespace/operators)
			LSTMActsForward<Dtype> << <CAFFE_GET_BLOCKS(X_count), CAFFE_CUDA_NUM_THREADS >> >(
				X_count, hidden_dim_, X, X_acts);
			CUDA_POST_KERNEL_CHECK;
			// NOLINT_NEXT_LINE(whitespace/operators)
			LSTMUnitForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, hidden_dim_, C_prev, X_acts, C, H);
			CUDA_POST_KERNEL_CHECK;
		}
	}

	template <typename Dtype>
	__global__ void LSTMUnitBackward(const int nthreads, const int dim,
		const Dtype* C_prev, const Dtype* X, const Dtype* C, const Dtype* H,
		const Dtype* C_diff, const Dtype* H_diff,
		Dtype* C_prev_diff, Dtype* X_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int n = index / dim;
			const int d = index % dim;
			const Dtype* X_offset = X + 4 * dim * n;
			const Dtype i = X_offset[d];
			const Dtype f = X_offset[1 * dim + d];
			const Dtype o = X_offset[2 * dim + d];
			const Dtype g = X_offset[3 * dim + d];
			const Dtype c_prev = C_prev[index];
			const Dtype c = C[index];
			const Dtype tanh_c = tanh(c);
			Dtype* c_prev_diff = C_prev_diff + index;
			Dtype* X_diff_offset = X_diff + 4 * dim * n;
			Dtype* i_diff = X_diff_offset + d;
			Dtype* f_diff = X_diff_offset + 1 * dim + d;
			Dtype* o_diff = X_diff_offset + 2 * dim + d;
			Dtype* g_diff = X_diff_offset + 3 * dim + d;
			const Dtype c_term_diff =
				C_diff[index] + H_diff[index] * o * (1 - tanh_c * tanh_c);
			*c_prev_diff = c_term_diff * f;
			*i_diff = c_term_diff * g;
			*f_diff = c_term_diff * c_prev;
			*o_diff = H_diff[index] * tanh_c;
			*g_diff = c_term_diff * i;
		}
	}

	template <typename Dtype>
	__global__ void LSTMActsBackward(const int nthreads, const int dim,
		const Dtype* X_acts, const Dtype* X_acts_diff, Dtype* X_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int x_dim = 4 * dim;
			const int d = index % x_dim;
			const Dtype X_act = X_acts[index];
			if (d < 3 * dim) {
				X_diff[index] = X_acts_diff[index] * X_act * (Dtype(1) - X_act);
			}
			else {
				X_diff[index] = X_acts_diff[index] * (Dtype(1) - X_act * X_act);
			}
		}
	}

	template <typename Dtype>
	void LSTMUnitLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		bool prop_down = false;
		for (int i = 0; i < propagate_down.size(); i++)
		{
			if (propagate_down[i])
			{
				prop_down = true;
			}
		}
		if (!prop_down)
		{
			return;
		}

		const int count = top[1]->count();
		const int X_count = bottom[1]->count();
		for (int i = 0; i < this->input_num_; i++)
		{
			const Dtype* C_prev = bottom[i * 2]->gpu_data();
			const Dtype* X = bottom[i * 2 + 1]->gpu_data();
			const Dtype* C = top[i * 2]->gpu_data();
			const Dtype* H = top[i * 2 + 1]->gpu_data();
			const Dtype* C_diff = top[i * 2]->gpu_diff();
			const Dtype* H_diff = top[i * 2 + 1]->gpu_diff();
			Dtype* C_prev_diff = bottom[i * 2]->mutable_gpu_diff();
			Dtype* X_diff = bottom[i * 2 + 1]->mutable_gpu_diff();

			Dtype* X_acts = X_acts_[i]->mutable_gpu_data();
			Dtype* X_acts_diff = X_acts_[i]->mutable_gpu_diff();

			LSTMActsForward<Dtype> << <CAFFE_GET_BLOCKS(X_count), CAFFE_CUDA_NUM_THREADS >> >(
				X_count, hidden_dim_, X, X_acts);
			CUDA_POST_KERNEL_CHECK;
			LSTMUnitBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
				<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(count, hidden_dim_,
				C_prev, X_acts, C, H, C_diff, H_diff, C_prev_diff, X_acts_diff);
			CUDA_POST_KERNEL_CHECK;
			LSTMActsBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
				<< <CAFFE_GET_BLOCKS(X_count), CAFFE_CUDA_NUM_THREADS >> >(
				X_count, hidden_dim_, X_acts, X_acts_diff, X_diff);
			CUDA_POST_KERNEL_CHECK;
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(LSTMUnitLayer);

}  // namespace caffe