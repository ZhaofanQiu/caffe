﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/9/27
** desc： ExtractFrame layer
*********************************************************************************/

#include <vector>

#include "caffe/video_layers.hpp"

namespace caffe {

	template <typename Dtype>
	void ExtractFrameLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();

		int length = bottom[0]->shape(2);
		int outer = bottom[0]->count(0, 2);
		int inner = bottom[0]->count(3);
		caffe_gpu_set(top[0]->count(), static_cast<Dtype>(0), top_data);
		for (int o = 0; o < outer; o++)
		{
			caffe_copy(inner, bottom_data + frame_id_ * inner, top_data);
			bottom_data += inner * length;
			top_data += inner;
		}
	}

	template <typename Dtype>
	void ExtractFrameLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

		int length = bottom[0]->shape(2);
		int outer = bottom[0]->count(0, 2);
		int inner = bottom[0]->count(3);
		if (propagate_down[0]) {
			caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0), bottom_diff);
			for (int o = 0; o < outer; o++)
			{
				caffe_copy(inner, top_diff, bottom_diff + frame_id_ * inner);
				bottom_diff += inner * length;
				top_diff += inner;
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(ExtractFrameLayer);
}  // namespace caffe
