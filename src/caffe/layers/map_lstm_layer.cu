﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/10/19
** desc： MapLSTMLayer layer
*********************************************************************************/

#include <algorithm>
#include <cmath>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/video_layers.hpp"

namespace caffe {

	template <typename Dtype>
	void MapLSTMLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int num = bottom[0]->shape(0);
		const int channels = bottom[0]->shape(1);

		const int outer = bottom[0]->count(0, 2);
		const int length = bottom[0]->shape(2);
		const int inner = bottom[0]->count(3);

		conv_->blobs()[0]->ShareData(*(blobs_[0]));
		if (bias_term_)
		{
			conv_->blobs()[1]->ShareData(*(blobs_[1]));
		}
		// 1. copy bottom to X_.
		const Dtype* bottom_data = bottom[0]->gpu_data();
		for (int o = 0; o < outer; ++o)
		{
			for (int l = 0; l < T_; ++l)
			{
				Dtype* X_data = X_[l]->mutable_gpu_data();
				caffe_copy(inner, bottom_data, X_data + o * inner);
				bottom_data += inner;
			}
		}
		// For all sequence run lstm.
		for (int t = 0; t < T_; t++)
		{
			//2. concat x & h_t-1.
			vector<Blob<Dtype>*> concat_bottom(2, NULL);
			concat_bottom[0] = X_[t].get();
			if (t == 0)
			{
				concat_bottom[1] = zero_memory_.get();
			}
			else
			{
				concat_bottom[1] = H_1_[t - 1].get();
			}

			const vector<Blob<Dtype>*> concat_top(1, XH_[t].get());
			concat_->Forward(concat_bottom, concat_top);

			//3. forward gate.
			const vector<Blob<Dtype>*> conv_bottom(1, XH_[t].get());
			const vector<Blob<Dtype>*> conv_top(1, G_[t].get());
			conv_->Forward(conv_bottom, conv_top);

			//4. LSTM Unit.
			vector<Blob<Dtype>*> lstm_bottom(2, NULL);
			if (t == 0)
			{
				lstm_bottom[0] = zero_memory_.get();
			}
			else
			{
				lstm_bottom[0] = C_[t - 1].get();
			}
			lstm_bottom[1] = G_[t].get();

			vector<Blob<Dtype>*> lstm_top{
				C_[t].get(),
				H_[t].get()
			};
			lstm_unit_->Forward(lstm_bottom, lstm_top);
			// 5 split
			const vector<Blob<Dtype>*> split_bottom(1, H_[t].get());
			const vector<Blob<Dtype>*> split_top{ H_1_[t].get(), H_2_[t].get() };
			split_h_->Forward(split_bottom, split_top);
		}
		//6. copy top.
		Dtype* top_data = top[0]->mutable_gpu_data();
		const int out_outer = top[0]->count(0, 2);
		for (int o = 0; o < out_outer; ++o)
		{
			for (int l = 0; l < T_; ++l)
			{
				const Dtype* X_data = H_2_[l]->gpu_data();
				caffe_copy(inner, X_data + o * inner, top_data);
				top_data += inner;
			}
		}
	}

	template <typename Dtype>
	void MapLSTMLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		if (!propagate_down[0])
		{
			return;
		}

		const int num = bottom[0]->shape(0);
		const int channels = bottom[0]->shape(1);

		const int outer = bottom[0]->count(0, 2);
		const int out_outer = top[0]->count(0, 2);
		const int length = bottom[0]->shape(2);
		const int inner = bottom[0]->count(3);;

		conv_->blobs()[0]->ShareData(*(blobs_[0]));
		if (bias_term_)
		{
			conv_->blobs()[1]->ShareData(*(blobs_[1]));
		}
		//6. copy top.
		const Dtype* top_data = top[0]->gpu_diff();
		for (int o = 0; o < out_outer; ++o)
		{
			for (int l = 0; l < T_; ++l)
			{
				Dtype* X_data = H_2_[l]->mutable_gpu_diff();
				caffe_copy<Dtype>(inner, top_data, X_data + o * inner);
				top_data += inner;
			}
		}

		// For all sequence run lstm.
		for (int t = T_ - 1; t >= 0; t++)
		{
			// 5 split
			const vector<Blob<Dtype>*> split_bottom(1, H_[t].get());
			const vector<Blob<Dtype>*> split_top{ H_1_[t].get(), H_2_[t].get() };
			split_h_->Backward(split_top, vector<bool>(1, true), split_bottom);

			//4. LSTM Unit.
			vector<Blob<Dtype>*> lstm_bottom(2, NULL);
			if (t == 0)
			{
				lstm_bottom[0] = zero_memory_.get();
			}
			else
			{
				lstm_bottom[0] = C_[t - 1].get();
			}
			lstm_bottom[1] = G_[t].get();
			vector<Blob<Dtype>*> lstm_top{
				C_[t].get(),
				H_[t].get()
			};
			lstm_unit_->Backward(lstm_top, vector<bool>(2, true), lstm_bottom);

			//3. forward gate.
			const vector<Blob<Dtype>*> conv_bottom(1, XH_[t].get());
			const vector<Blob<Dtype>*> conv_top(1, G_[t].get());
			conv_->Backward(conv_top, vector<bool>(1, true), conv_bottom);

			//2. concat x & h_t-1.
			vector<Blob<Dtype>*> concat_bottom(2, NULL);
			concat_bottom[0] = X_[t].get();
			if (t == 0)
			{
				concat_bottom[1] = zero_memory_.get();
			}
			else
			{
				concat_bottom[1] = H_1_[t - 1].get();
			}

			const vector<Blob<Dtype>*> concat_top(1, XH_[t].get());
			concat_->Backward(concat_top, vector<bool>(2, true), concat_bottom);
		}
		//6. copy top.
		Dtype* bottom_data = bottom[0]->mutable_gpu_diff();
		for (int o = 0; o < outer; ++o)
		{
			for (int l = 0; l < T_; ++l)
			{
				const Dtype* X_data = X_[l]->gpu_diff();
				caffe_copy(inner, X_data + o * inner, bottom_data);
				bottom_data += inner;
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(MapLSTMLayer);
}  // namespace caffe