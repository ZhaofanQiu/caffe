#include "hip/hip_runtime.h"
﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/9/14
** desc： Video switch layer
*********************************************************************************/

#include <vector>

#include "caffe/video_layers.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void video_switch_forward_kernel(const int n, const int num,
		const int channels, const int length, const int inner,
		const Dtype* src, Dtype* dest) {
		CUDA_KERNEL_LOOP(index, n) {
			int l = index % length;
			int c = (index / length) % channels;
			int n = index / length / channels;
			int res_offset = ((n * length + l) * channels + c) * inner;
			int des_offset = ((n * channels + c) * length + l) * inner;
			for (int i = 0; i < inner; ++i) {
				dest[des_offset + i] = src[res_offset + i];
			}
		}
	}

	template <typename Dtype>
	void video_switch_forward_gpu(const Dtype* res, Dtype* des,
		const vector<int> res_shape, const vector<int> des_shape,
		int frame_num, int inner)
	{
		const int outer = des_shape[0] * des_shape[1] * des_shape[2];

		// NOLINT_NEXT_LINE(whitespace/operators)
		video_switch_forward_kernel << <CAFFE_GET_BLOCKS(outer), CAFFE_CUDA_NUM_THREADS >> >(
			outer, des_shape[0], des_shape[1],
			des_shape[2], inner, res, des);
	}

	template <typename Dtype>
	__global__ void video_switch_backward_kernel(const int n, const int num,
		const int channels, const int length, const int inner,
		const Dtype* src, Dtype* dest) {
		CUDA_KERNEL_LOOP(index, n) {
			int l = index % length;
			int c = (index / length) % channels;
			int n = index / length / channels;
			int res_offset = ((n * channels + c) * length + l) * inner;
			int des_offset = ((n * length + l) * channels + c) * inner;
			for (int i = 0; i < inner; ++i) {
				dest[des_offset + i] = src[res_offset + i];
			}
		}
	}

	template <typename Dtype>
	void video_switch_backward_gpu(const Dtype* res, Dtype* des,
		const vector<int> res_shape, const vector<int> des_shape,
		int frame_num, int inner)
	{
		const int outer = res_shape[0] * res_shape[1] * res_shape[2];

		// NOLINT_NEXT_LINE(whitespace/operators)
		video_switch_backward_kernel << <CAFFE_GET_BLOCKS(outer), CAFFE_CUDA_NUM_THREADS >> >(
			outer, res_shape[0], res_shape[1],
			res_shape[2], inner, res, des);
	}

	template <typename Dtype>
	void VideoSwitchLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();

		if (this->to_video_)
		{
			video_switch_forward_gpu(bottom_data, top_data,
				bottom[0]->shape(), top[0]->shape(), frame_num_, top[0]->count(3));
		}
		else
		{
			video_switch_backward_gpu(bottom_data, top_data,
				bottom[0]->shape(), top[0]->shape(), frame_num_, bottom[0]->count(3));
		}
	}

	template <typename Dtype>
	void VideoSwitchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

		if (propagate_down[0]) {
			if (this->to_video_)
			{
				video_switch_backward_gpu(top_diff, bottom_diff,
					top[0]->shape(), bottom[0]->shape(), frame_num_, top[0]->count(3));
			}
			else
			{
				video_switch_forward_gpu(top_diff, bottom_diff,
					top[0]->shape(), bottom[0]->shape(), frame_num_, bottom[0]->count(3));
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(VideoSwitchLayer);
}  // namespace caffe
