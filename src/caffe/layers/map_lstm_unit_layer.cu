#include "hip/hip_runtime.h"
﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/10/19
** desc： MapLSTMUnit layer
*********************************************************************************/

#include <algorithm>
#include <cmath>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/video_layers.hpp"

namespace caffe {

	template <typename Dtype>
	__device__ Dtype sigmoid(const Dtype x) {
		return Dtype(1) / (Dtype(1) + exp(-x));
	}

	template <typename Dtype>
	__device__ Dtype tanh(const Dtype x) {
		return Dtype(2) * sigmoid(Dtype(2) * x) - Dtype(1);
	}

	template <typename Dtype>
	__global__ void MapLSTMActsForward(const int nthreads, const int dim,
		const Dtype* X, Dtype* X_acts) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int x_dim = 4 * dim;
			const int d = index % x_dim;
			if (d < 3 * dim) {
				X_acts[index] = sigmoid(X[index]);
			}
			else {
				X_acts[index] = tanh(X[index]);
			}
		}
	}

	template <typename Dtype>
	__global__ void MapLSTMUnitForward(const int nthreads, const int dim,
		const Dtype* C_prev, const Dtype* X,
		Dtype* C, Dtype* H) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int n = index / dim;
			const int d = index % dim;
			const Dtype* X_offset = X + 4 * dim * n;
			const Dtype i = X_offset[d];
			const Dtype f = X_offset[1 * dim + d];
			const Dtype o = X_offset[2 * dim + d];
			const Dtype g = X_offset[3 * dim + d];
			const Dtype c_prev = C_prev[index];
			const Dtype c = f * c_prev + i * g;
			C[index] = c;
			const Dtype tanh_c = tanh(c);
			H[index] = o * tanh_c;
		}
	}

	template <typename Dtype>
	void MapLSTMUnitLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int outer = bottom[0]->shape(0);
		const int channels = bottom[0]->shape(1);
		const int inner = bottom[0]->count(1);

		const Dtype* C_prev = bottom[0]->gpu_data();
		const Dtype* X = bottom[1]->gpu_data();
		Dtype* C = top[0]->mutable_gpu_data();
		Dtype* H = top[1]->mutable_gpu_data();
		Dtype* X_acts = X_acts_->mutable_gpu_data();
		// NOLINT_NEXT_LINE(whitespace/operators)
		MapLSTMActsForward<Dtype> << <CAFFE_GET_BLOCKS(outer * inner * 4), CAFFE_CUDA_NUM_THREADS >> >(
			outer * inner * 4, inner * 4, X, X_acts);
		CUDA_POST_KERNEL_CHECK;
		// NOLINT_NEXT_LINE(whitespace/operators)
		MapLSTMUnitForward<Dtype> << <CAFFE_GET_BLOCKS(outer * inner), CAFFE_CUDA_NUM_THREADS >> >(
			outer * inner, inner, C_prev, X_acts, C, H);
		CUDA_POST_KERNEL_CHECK;
	}

	template <typename Dtype>
	__global__ void MapLSTMUnitBackward(const int nthreads, const int dim,
		const Dtype* C_prev, const Dtype* X, const Dtype* C, const Dtype* H,
		const Dtype* C_diff, const Dtype* H_diff,
		Dtype* C_prev_diff, Dtype* X_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int n = index / dim;
			const int d = index % dim;
			const Dtype* X_offset = X + 4 * dim * n;
			const Dtype i = X_offset[d];
			const Dtype f = X_offset[1 * dim + d];
			const Dtype o = X_offset[2 * dim + d];
			const Dtype g = X_offset[3 * dim + d];
			const Dtype c_prev = C_prev[index];
			const Dtype c = C[index];
			const Dtype tanh_c = tanh(c);
			Dtype* c_prev_diff = C_prev_diff + index;
			Dtype* X_diff_offset = X_diff + 4 * dim * n;
			Dtype* i_diff = X_diff_offset + d;
			Dtype* f_diff = X_diff_offset + 1 * dim + d;
			Dtype* o_diff = X_diff_offset + 2 * dim + d;
			Dtype* g_diff = X_diff_offset + 3 * dim + d;
			const Dtype c_term_diff =
				C_diff[index] + H_diff[index] * o * (1 - tanh_c * tanh_c);
			*c_prev_diff = c_term_diff * f;
			*i_diff = c_term_diff * g;
			*f_diff = c_term_diff * c_prev;
			*o_diff = H_diff[index] * tanh_c;
			*g_diff = c_term_diff * i;
		}
	}

	template <typename Dtype>
	__global__ void MapLSTMActsBackward(const int nthreads, const int dim,
		const Dtype* X_acts, const Dtype* X_acts_diff, Dtype* X_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			const int x_dim = 4 * dim;
			const int d = index % x_dim;
			const Dtype X_act = X_acts[index];
			if (d < 3 * dim) {
				X_diff[index] = X_acts_diff[index] * X_act * (Dtype(1) - X_act);
			}
			else {
				X_diff[index] = X_acts_diff[index] * (Dtype(1) - X_act * X_act);
			}
		}
	}

	template <typename Dtype>
	void MapLSTMUnitLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		if (!propagate_down[0] && !propagate_down[1])
		{
			return;
		}

		const int outer = bottom[0]->shape(0);
		const int channels = bottom[0]->shape(1);
		const int inner = bottom[0]->count(1);

		const Dtype* C_prev = bottom[0]->gpu_data();
		const Dtype* X = bottom[1]->gpu_data();
		const Dtype* C = top[0]->gpu_data();
		const Dtype* H = top[1]->gpu_data();
		const Dtype* C_diff = top[0]->gpu_diff();
		const Dtype* H_diff = top[1]->gpu_diff();
		Dtype* C_prev_diff = bottom[0]->mutable_gpu_diff();
		Dtype* X_diff = bottom[1]->mutable_gpu_diff();

		Dtype* X_acts = X_acts_->mutable_gpu_data();
		Dtype* X_acts_diff = X_acts_->mutable_gpu_diff();

		MapLSTMActsForward<Dtype> << <CAFFE_GET_BLOCKS(outer * inner * 4), CAFFE_CUDA_NUM_THREADS >> >(
			outer * inner * 4, inner * 4, X, X_acts);
		CUDA_POST_KERNEL_CHECK;
		MapLSTMUnitBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
			<< <CAFFE_GET_BLOCKS(outer * inner), CAFFE_CUDA_NUM_THREADS >> >(outer * inner, inner,
			C_prev, X_acts, C, H, C_diff, H_diff, C_prev_diff, X_acts_diff);
		CUDA_POST_KERNEL_CHECK;
		MapLSTMActsBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
			<< <CAFFE_GET_BLOCKS(outer * inner * 4), CAFFE_CUDA_NUM_THREADS >> >(
			outer * inner * 4, inner * 4, X_acts, X_acts_diff, X_diff);
		CUDA_POST_KERNEL_CHECK;
	}

	INSTANTIATE_LAYER_GPU_FUNCS(MapLSTMUnitLayer);
}  // namespace caffe