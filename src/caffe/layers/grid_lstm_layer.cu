#include "hip/hip_runtime.h"
﻿
/********************************************************************************
** Copyright(c) 2015 USTC & MSRA All Rights Reserved.
** auth： Zhaofan Qiu
** mail： zhaofanqiu@gmail.com
** date： 2015/9/15
** desc： GirdLSTMLayer layer
*********************************************************************************/

#include <algorithm>
#include <cmath>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/video_layers.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void Blob2Xd(int n, const Dtype* blob, int d, 
		int num_seq, Dtype* xd) {
		CUDA_KERNEL_LOOP(index, n) {
			xd[index] = blob[index * num_seq + d];
		}
	}

	template <typename Dtype>
	__global__ void Xd2Blob(int n, const Dtype* xd, int d,
		int num_seq, Dtype* blob) {
		CUDA_KERNEL_LOOP(index, n) {
			blob[index * num_seq + d] = xd[index];
		}
	}

	template <typename Dtype>
	void GridLSTMLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int num = bottom[0]->shape(0);
		const int channels = bottom[0]->shape(1);
		// 1. copy bottom to X_.
		const Dtype* bottom_data = bottom[0]->gpu_data();
		for (int d = 0; d < num_seq_; d++)
		{
			Dtype* X_data = X_[d]->mutable_gpu_data();
			Blob2Xd<Dtype> << <CAFFE_GET_BLOCKS(num * channels), CAFFE_CUDA_NUM_THREADS >> >(
				num * channels, bottom_data, d, num_seq_, X_data);
		}
		// 1.5 split
		for (int i = 0; i < num_seq_; i++)
		{
			vector<Blob<Dtype>*> split2_bottom_vec(1, X_[i].get());
			vector<Blob<Dtype>*> split2_top_vec(3, NULL);
			split2_top_vec[0] = X_1_[i].get();
			split2_top_vec[1] = X_2_[i].get();
			split2_top_vec[2] = X_3_[i].get();
			split_x_->Forward(split2_bottom_vec, split2_top_vec);
		}
		// For all sequence run lstm1.
		for (int d = 0; d < num_seq_; d++)
		{
			int dp = order_[d];
			//2. concat x & h_t-1.
			vector<Blob<Dtype>*> concat_bottom_vec(grid_dim_ + 1, NULL);
			concat_bottom_vec[0] = X_1_[dp].get();
			for (int i = 0; i < grid_dim_; i++)
			{
				if (link_idx_[dp][i] < 0)
				{
					concat_bottom_vec[1 + i] = zero_memory_.get();
				}
				else
				{
					concat_bottom_vec[1 + i] = H_i_1_[link_idx_[dp][i]][i].get();
				}
			}
			vector<Blob<Dtype>*> concat_top_vec(1, XH_h_[dp].get());
			concat_h_->Forward(concat_bottom_vec, concat_top_vec);
			//2.5. split xh_h.
			vector<Blob<Dtype>*> split_xh_bottom_vec(1, XH_h_[dp].get());
			vector<Blob<Dtype>*> split_xh_top_vec(grid_dim_, NULL);
			for (int i = 0; i < grid_dim_; i++)
			{
				split_xh_top_vec[i] = XH_h_k_[dp][i].get();
			}
			split_xh_h_->Forward(split_xh_bottom_vec, split_xh_top_vec);
			//3. forward gate.
			for (int i = 0; i < grid_dim_; i++)
			{
				vector<Blob<Dtype>*> ip_bottom_vec(1, XH_h_k_[dp][i].get());
				vector<Blob<Dtype>*> ip_top_vec(1, G_h_[dp][i].get());
				ip_xh_h_[i]->Forward(ip_bottom_vec, ip_top_vec);
			}
			//4. LSTM Unit 1.
			vector<Blob<Dtype>*> lstm_bottom_vec(grid_dim_ * 2, NULL);
			vector<Blob<Dtype>*> lstm_top_vec(grid_dim_ * 2, NULL);
			for (int i = 0; i < grid_dim_; i++)
			{
				if (link_idx_[dp][i] < 0)
				{
					lstm_bottom_vec[i * 2] = zero_memory_.get();
				}
				else
				{
					lstm_bottom_vec[i * 2] = C_i_[link_idx_[dp][i]][i].get();
				}
				lstm_bottom_vec[i * 2 + 1] = G_h_[dp][i].get();
				lstm_top_vec[i * 2] = C_i_[dp][i].get();
				lstm_top_vec[i * 2 + 1] = H_i_[dp][i].get();
			}
			lstm_unit_h_->Forward(lstm_bottom_vec, lstm_top_vec);
			// 4.5 split
			for (int i = 0; i < grid_dim_; i++)
			{
				vector<Blob<Dtype>*> split1_bottom_vec(1, H_i_[dp][i].get());
				vector<Blob<Dtype>*> split1_top_vec(2, NULL);
				split1_top_vec[0] = H_i_1_[dp][i].get();
				split1_top_vec[1] = H_i_2_[dp][i].get();
				split_h_->Forward(split1_bottom_vec, split1_top_vec);
			}
		}
		// For all sequence run lstm2.
		for (int d = 0; d < num_seq_; d++)
		{
			int dp = order_[d];
			//5. concat x & h_t
			vector<Blob<Dtype>*> concat_bottom_vec(1 + grid_dim_, NULL);
			concat_bottom_vec[0] = X_2_[dp].get();
			for (int i = 0; i < grid_dim_; i++)
			{
				concat_bottom_vec[1 + i] = H_i_2_[dp][i].get();
			}
			vector<Blob<Dtype>*> concat_top_vec(1, XH_x_[dp].get());
			concat_x_->Forward(concat_bottom_vec, concat_top_vec);
			//6. forward gate.
			vector<Blob<Dtype>*> ip_bottom_vec(1, XH_x_[dp].get());
			vector<Blob<Dtype>*> ip_top_vec(1, G_x_[dp].get());
			ip_xh_x_->Forward(ip_bottom_vec, ip_top_vec);
			//7. LSTM Unit 2.
			vector<Blob<Dtype>*> lstm_bottom_vec(2, NULL);
			vector<Blob<Dtype>*> lstm_top_vec(2, NULL);
			lstm_bottom_vec[0] = X_3_[dp].get();
			lstm_bottom_vec[1] = G_x_[dp].get();
			lstm_top_vec[0] = X_c_[dp].get();
			lstm_top_vec[1] = X_h_[dp].get();
			lstm_unit_x_->Forward(lstm_bottom_vec, lstm_top_vec);
		}
		//8. copy top.
		Dtype* top_data = top[0]->mutable_gpu_data();
		for (int d = 0; d < num_seq_; d++)
		{
			const Dtype* X_data = X_h_[d]->gpu_data();
			Xd2Blob<Dtype> << <CAFFE_GET_BLOCKS(num * channels), CAFFE_CUDA_NUM_THREADS >> >(
				num * channels, X_data, d, num_seq_, top_data);
		}
	}

	template <typename Dtype>
	void GridLSTMLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		const int num = bottom[0]->shape(0);
		const int channels = bottom[0]->shape(1);
		
		//8. copy top.
		const Dtype* top_diff = top[0]->gpu_diff();
		for (int d = 0; d < num_seq_; d++)
		{
			Dtype* X_diff = X_h_[d]->mutable_gpu_diff();
			Blob2Xd<Dtype> << <CAFFE_GET_BLOCKS(num * channels), CAFFE_CUDA_NUM_THREADS >> >(
				num * channels, top_diff, d, num_seq_, X_diff);
		}
		// For all sequence run lstm2.
		for (int d = 0; d < num_seq_; d++)
		{
			int dp = order_[num_seq_ - 1 - d];
			//7. LSTM Unit 2.
			vector<Blob<Dtype>*> lstm_bottom_vec(2, NULL);
			vector<Blob<Dtype>*> lstm_top_vec(2, NULL);
			vector<bool> lstm_prop(2, true);
			lstm_bottom_vec[0] = X_3_[dp].get();
			lstm_bottom_vec[1] = G_x_[dp].get();
			lstm_top_vec[0] = X_c_[dp].get();
			lstm_top_vec[1] = X_h_[dp].get();
			lstm_unit_x_->Backward(lstm_top_vec, lstm_prop, lstm_bottom_vec);
			//6. forward gate.
			vector<Blob<Dtype>*> ip_bottom_vec(1, XH_x_[dp].get());
			vector<Blob<Dtype>*> ip_top_vec(1, G_x_[dp].get());
			vector<bool> ip_prop(1, true);
			ip_xh_x_->Backward(ip_top_vec, ip_prop, ip_bottom_vec);
			//5. concat x & h_t
			vector<Blob<Dtype>*> concat_bottom_vec(1 + grid_dim_, NULL);
			vector<bool> concat_prop(1 + grid_dim_, true);
			concat_bottom_vec[0] = X_2_[dp].get();
			concat_prop[0] = true;
			for (int i = 0; i < grid_dim_; i++)
			{
				concat_bottom_vec[1 + i] = H_i_2_[dp][i].get();
				concat_prop[1 + i] = true;
			}
			vector<Blob<Dtype>*> concat_top_vec(1, XH_x_[dp].get());
			concat_x_->Backward(concat_top_vec, concat_prop, concat_bottom_vec);
		}

		// For all sequence run lstm1.
		for (int d = 0; d < num_seq_; d++)
		{
			int dp = order_[num_seq_ - 1 - d];
			// 4.5 split
			for (int i = 0; i < grid_dim_; i++)
			{
				vector<Blob<Dtype>*> split1_bottom_vec(1, H_i_[dp][i].get());
				vector<bool> split1_prop(1, true);
				vector<Blob<Dtype>*> split1_top_vec(2, NULL);
				split1_top_vec[0] = H_i_1_[dp][i].get();
				split1_top_vec[1] = H_i_2_[dp][i].get();
				split_h_->Backward(split1_top_vec, split1_prop, split1_bottom_vec);
			}
			//4. LSTM Unit 1.
			vector<Blob<Dtype>*> lstm_bottom_vec(grid_dim_ * 2, NULL);
			vector<bool> lstm_prop(grid_dim_ * 2, true);
			vector<Blob<Dtype>*> lstm_top_vec(grid_dim_ * 2, NULL);
			for (int i = 0; i < grid_dim_; i++)
			{
				if (link_idx_[dp][i] < 0)
				{
					lstm_bottom_vec[i * 2] = zero_memory_.get();
					lstm_prop[i * 2] = false;
				}
				else
				{
					lstm_bottom_vec[i * 2] = C_i_[link_idx_[dp][i]][i].get();
					lstm_prop[i * 2] = true;
				}
				lstm_bottom_vec[i * 2 + 1] = G_h_[dp][i].get();
				lstm_prop[i * 2 + 1] = true;
				lstm_top_vec[i * 2] = C_i_[dp][i].get();
				lstm_top_vec[i * 2 + 1] = H_i_[dp][i].get();
			}
			lstm_unit_h_->Backward(lstm_top_vec, lstm_prop, lstm_bottom_vec);
			//3. forward gate.
			for (int i = 0; i < grid_dim_; i++)
			{
				vector<Blob<Dtype>*> ip_bottom_vec(1, XH_h_k_[dp][i].get());
				vector<bool> ip_prop(1, true);
				vector<Blob<Dtype>*> ip_top_vec(1, G_h_[dp][i].get());
				ip_xh_h_[i]->Backward(ip_top_vec, ip_prop, ip_bottom_vec);
			}
			//2.5. split xh_h.
			vector<Blob<Dtype>*> split_xh_bottom_vec(1, XH_h_[dp].get());
			vector<bool> split_xh_prop(1, true);
			vector<Blob<Dtype>*> split_xh_top_vec(grid_dim_, NULL);
			for (int i = 0; i < grid_dim_; i++)
			{
				split_xh_top_vec[i] = XH_h_k_[dp][i].get();
			}
			split_xh_h_->Backward(split_xh_top_vec, split_xh_prop, split_xh_bottom_vec);
			//2. concat x & h_t-1.
			vector<Blob<Dtype>*> concat_bottom_vec(1 + grid_dim_, NULL);
			concat_bottom_vec[0] = X_1_[dp].get();
			vector<bool> concat_prop(1 + grid_dim_, true);
			concat_prop[0] = true;
			for (int i = 0; i < grid_dim_; i++)
			{
				if (link_idx_[dp][i] < 0)
				{
					concat_bottom_vec[1 + i] = zero_memory_.get();
					concat_prop[1 + i] = false;
				}
				else
				{
					concat_bottom_vec[1 + i] = H_i_1_[link_idx_[dp][i]][i].get();
					concat_prop[1 + i] = true;
				}
			}
			vector<Blob<Dtype>*> concat_top_vec(1, XH_h_[dp].get());
			concat_h_->Backward(concat_top_vec, concat_prop, concat_bottom_vec);
		}
		// 1.5 split
		for (int i = 0; i < num_seq_; i++)
		{
			vector<Blob<Dtype>*> split2_bottom_vec(1, X_[i].get());
			vector<bool> split2_prop(1, true);
			vector<Blob<Dtype>*> split2_top_vec(3, NULL);
			split2_top_vec[0] = X_1_[i].get();
			split2_top_vec[1] = X_2_[i].get();
			split2_top_vec[2] = X_3_[i].get();
			split_x_->Backward(split2_top_vec, split2_prop, split2_bottom_vec);
		}
		// 1. copy bottom to X_.
		if (propagate_down[0])
		{
			Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
			for (int d = 0; d < num_seq_; d++)
			{
				const Dtype* X_diff = X_[d]->gpu_diff();
				Xd2Blob<Dtype> << <CAFFE_GET_BLOCKS(num * channels), CAFFE_CUDA_NUM_THREADS >> >(
					num * channels, X_diff, d, num_seq_, bottom_diff);
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(GridLSTMLayer);
}  // namespace caffe